#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>

#include <iostream>
#include <vector>
#include <cmath>
#include <algorithm> 
#include <random>
#include <chrono>
#include <stdlib.h> 
#include <iterator>
#include <random>
#include <chrono>
#include <fstream>
#include <bit>
#include <bitset>
#include <cstdint>
#include <stack>
#include <utility>
#include <map>
#include <ctime>
#include <omp.h>

using std::vector;
using std::cin;
using std::cout;
using std::endl;
using std::bitset;
using std::map;
using std::stack;
using std::pair;
using std::make_pair;

void time_schedule(int &free_time, int start, int end){
    for (int i = 0; i < 24; i++){
        if (i >= start && i < end){
            free_time  |= (1 << i);
        }
        else if (start > end && (i >= start || i < end)){
            free_time |= (1 << i);
        }
        else if(start == end){
            free_time |= (1 << i);
        }
    }
}

struct Movie{
    int start;
    int end;
    int genre;
};


struct exaustiva 
{  
    int n_movies;
    int n_genres;
    int* movies_by_genre;
    int* movies_time;
    int* movies_genre;


    exaustiva(int n_movies_, int n_genres_, int* movies_by_genre_, int* movies_time_, int* movies_genre_) : 
    n_movies(n_movies_), n_genres(n_genres_), movies_by_genre(movies_by_genre_), movies_time(movies_time_), movies_genre(movies_genre_) {}
    
    
    __host__ __device__
    int operator()(const int& config) {

        int free_time = 0;
        int n_movie = 0;
        int movies_by_genre_aux[20];
        
        for (int i = 0; i < n_genres; i++){
            movies_by_genre_aux[i] = *(movies_by_genre+i);
        }
        
        for (int i = 0; i < n_movies; i++){
            if (config & (1 << i)){
                if (movies_by_genre_aux[movies_genre[i]-1] > 0){
                    int a_time = free_time & *(movies_time + i);
                    if (((a_time) != 0)) return false;
                    movies_by_genre_aux[movies_genre[i]-1]--;
                    free_time = free_time | *(movies_time + i);
                    n_movie++;
                }
            }
        
        }

        return n_movie;
    }
};



int main(){
    int n_movies, n_genres;
    cin >> n_movies >> n_genres;

    thrust::host_vector<int> movies_by_genre(n_genres, 0);
    Movie empty_movie = {0, 0, 0};
    vector<Movie> movies (n_movies, empty_movie);

    for (int i = 0; i < n_genres; i++){
        cin >> movies_by_genre[i];
    }

    for (int i = 0; i < n_movies; i++){
        Movie movie;
        cin >> movie.start >> movie.end >> movie.genre;
        if (movie.start == 0) movie.start = 24;
        if (movie.end == 0) movie.end = 24;
        if (movie.start < 0 || movie.end < 0) continue;

        movies[i] = movie;
    }

    thrust::host_vector<int> movies_genre(n_movies);
    thrust::host_vector<int> movies_times(n_movies); 


    for (int i = 0; i < n_movies; i++){
        movies_times[i] = 0;
        time_schedule(movies_times[i], movies[i].start-1, movies[i].end-1);
        movies_genre[i] = movies[i].genre;
    }

    thrust::device_vector<int> config_vector(pow(2, n_movies));
    thrust::sequence(config_vector.begin(), config_vector.end());
    thrust::device_vector<int> movies_genre_gpu(movies_genre);
    thrust::device_vector<int> movies_times_gpu(movies_times);
    thrust::device_vector<int> movies_by_genre_gpu(movies_by_genre);
    thrust::transform(config_vector.begin(), config_vector.end(), config_vector.begin(), exaustiva(n_movies, n_genres, raw_pointer_cast(movies_by_genre_gpu.data()), raw_pointer_cast(movies_times_gpu.data()), raw_pointer_cast(movies_genre_gpu.data())));
    thrust::host_vector<int> config_vector_final = config_vector;
    
    int max_count = 0;
    int iters = pow(2, n_movies);
    for (int i = 0; i < iters; i++){
        if (config_vector_cpu_final[i] > max_count){
            max_count = config_vector_final[i];
        }
    }

    cout << max_count << " movies watched." << endl;
}
